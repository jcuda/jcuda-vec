
#include <hip/hip_runtime.h>
/*
 * JCudaVec - Vector operations for JCuda 
 * http://www.jcuda.org
 *
 * Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
 */
 
extern "C"
__global__ void vec_set (size_t n, double *result, double  value)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = value;
    }
}


//=== Vector arithmetic ======================================================

extern "C"
__global__ void vec_add (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = x[id] + y[id];
    }
}


extern "C"
__global__ void vec_sub (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = x[id] - y[id];
    }
}


extern "C"
__global__ void vec_mul (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = x[id] * y[id];
    }
}


extern "C"
__global__ void vec_div (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = x[id] / y[id];
    }
}

extern "C"
__global__ void vec_negate (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = -x[id];
    }
}




//=== Vector-and-scalar arithmetic ===========================================

extern "C"
__global__ void vec_addScalar (size_t n, double *result, double  *x, double  y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = x[id] + y;
    }
}


extern "C"
__global__ void vec_subScalar (size_t n, double *result, double  *x, double  y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = x[id] - y;
    }
}


extern "C"
__global__ void vec_mulScalar (size_t n, double *result, double  *x, double  y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = x[id] * y;
    }
}


extern "C"
__global__ void vec_divScalar (size_t n, double *result, double  *x, double  y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = x[id] / y;
    }
}




extern "C"
__global__ void vec_scalarAdd (size_t n, double *result, double  x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = x + y[id];
    }
}


extern "C"
__global__ void vec_scalarSub (size_t n, double *result, double  x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = x - y[id];
    }
}


extern "C"
__global__ void vec_scalarMul (size_t n, double *result, double  x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = x * y[id];
    }
}


extern "C"
__global__ void vec_scalarDiv (size_t n, double *result, double  x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = x / y[id];
    }
}









//=== Vector comparison ======================================================

extern "C"
__global__ void vec_lt (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = (x[id] < y[id])?1.0:0.0;
    }
}


extern "C"
__global__ void vec_lte (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = (x[id] <= y[id])?1.0:0.0;
    }
}


extern "C"
__global__ void vec_eq (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = (x[id] == y[id])?1.0:0.0;
    }
}


extern "C"
__global__ void vec_gte (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = (x[id] >= y[id])?1.0:0.0;
    }
}


extern "C"
__global__ void vec_gt (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = (x[id] > y[id])?1.0:0.0;
    }
}



extern "C"
__global__ void vec_ne (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = (x[id] != y[id])?1.0:0.0;
    }
}




//=== Vector-and-scalar comparison ===========================================

extern "C"
__global__ void vec_ltScalar (size_t n, double *result, double  *x, double  y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = (x[id] < y)?1.0:0.0;
    }
}


extern "C"
__global__ void vec_lteScalar (size_t n, double *result, double  *x, double  y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = (x[id] <= y)?1.0:0.0;
    }
}


extern "C"
__global__ void vec_eqScalar (size_t n, double *result, double  *x, double  y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = (x[id] == y)?1.0:0.0;
    }
}


extern "C"
__global__ void vec_gteScalar (size_t n, double *result, double  *x, double  y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = (x[id] >= y)?1.0:0.0;
    }
}


extern "C"
__global__ void vec_gtScalar (size_t n, double *result, double  *x, double  y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = (x[id] > y)?1.0:0.0;
    }
}


extern "C"
__global__ void vec_neScalar (size_t n, double *result, double  *x, double  y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = (x[id] != y)?1.0:0.0;
    }
}











//=== Vector math (one argument) =============================================


// Calculate the arc cosine of the input argument.
extern "C"
__global__ void vec_acos (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = acos(x[id]);
    }
}


// Calculate the nonnegative arc hyperbolic cosine of the input argument.
extern "C"
__global__ void vec_acosh (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = acosh(x[id]);
    }
}


// Calculate the arc sine of the input argument.
extern "C"
__global__ void vec_asin (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = asin(x[id]);
    }
}


// Calculate the arc hyperbolic sine of the input argument.
extern "C"
__global__ void vec_asinh (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = asinh(x[id]);
    }
}


// Calculate the arc tangent of the input argument.
extern "C"
__global__ void vec_atan (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = atan(x[id]);
    }
}


// Calculate the arc hyperbolic tangent of the input argument.
extern "C"
__global__ void vec_atanh (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = atanh(x[id]);
    }
}


// Calculate the cube root of the input argument.
extern "C"
__global__ void vec_cbrt (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = cbrt(x[id]);
    }
}


// Calculate ceiling of the input argument.
extern "C"
__global__ void vec_ceil (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = ceil(x[id]);
    }
}


// Calculate the cosine of the input argument.
extern "C"
__global__ void vec_cos (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = cos(x[id]);
    }
}


// Calculate the hyperbolic cosine of the input argument.
extern "C"
__global__ void vec_cosh (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = cosh(x[id]);
    }
}


// Calculate the cosine of the input argument � p .
extern "C"
__global__ void vec_cospi (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = cospi(x[id]);
    }
}


// Calculate the complementary error function of the input argument.
extern "C"
__global__ void vec_erfc (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = erfc(x[id]);
    }
}


// Calculate the inverse complementary error function of the input argument.
extern "C"
__global__ void vec_erfcinv (size_t n, double *result, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = erfcinv(y[id]);
    }
}


// Calculate the scaled complementary error function of the input argument.
extern "C"
__global__ void vec_erfcx (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = erfcx(x[id]);
    }
}


// Calculate the error function of the input argument.
extern "C"
__global__ void vec_erf (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = erf(x[id]);
    }
}


// Calculate the inverse error function of the input argument.
extern "C"
__global__ void vec_erfinv (size_t n, double *result, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = erfinv(y[id]);
    }
}


// Calculate the base 10 exponential of the input argument.
extern "C"
__global__ void vec_exp10 (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = exp10(x[id]);
    }
}


// Calculate the base 2 exponential of the input argument.
extern "C"
__global__ void vec_exp2 (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = exp2(x[id]);
    }
}


// Calculate the base e exponential of the input argument.
extern "C"
__global__ void vec_exp (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = exp(x[id]);
    }
}


// Calculate the base e exponential of the input argument, minus 1.
extern "C"
__global__ void vec_expm1 (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = expm1(x[id]);
    }
}


// Calculate the absolute value of its argument.
extern "C"
__global__ void vec_fabs (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = fabs(x[id]);
    }
}


// Calculate the largest integer less than or equal to x.
extern "C"
__global__ void vec_floor (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = floor(x[id]);
    }
}


// Calculate the value of the Bessel function of the first kind of order 0 for the input argument.
extern "C"
__global__ void vec_j0 (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = j0(x[id]);
    }
}


// Calculate the value of the Bessel function of the first kind of order 1 for the input argument.
extern "C"
__global__ void vec_j1 (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = j1(x[id]);
    }
}


// Calculate the natural logarithm of the absolute value of the gamma function of the input argument.
extern "C"
__global__ void vec_lgamma (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = lgamma(x[id]);
    }
}


// Calculate the base 10 logarithm of the input argument.
extern "C"
__global__ void vec_log10 (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = log10(x[id]);
    }
}


// Calculate the value of l o g e ( 1 + x ) .
extern "C"
__global__ void vec_log1p (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = log1p(x[id]);
    }
}


// Calculate the base 2 logarithm of the input argument.
extern "C"
__global__ void vec_log2 (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = log2(x[id]);
    }
}


// Calculate the doubleing point representation of the exponent of the input argument.
extern "C"
__global__ void vec_logb (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = logb(x[id]);
    }
}


// Calculate the natural logarithm of the input argument.
extern "C"
__global__ void vec_log (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = log(x[id]);
    }
}


// Calculate the standard normal cumulative distribution function.
extern "C"
__global__ void vec_normcdf (size_t n, double *result, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = normcdf(y[id]);
    }
}


// Calculate the inverse of the standard normal cumulative distribution function.
extern "C"
__global__ void vec_normcdfinv (size_t n, double *result, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = normcdfinv(y[id]);
    }
}


// Calculate reciprocal cube root function.
extern "C"
__global__ void vec_rcbrt (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = rcbrt(x[id]);
    }
}


// Round input to nearest integer value in doubleing-point.
extern "C"
__global__ void vec_rint (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = rint(x[id]);
    }
}


// Round to nearest integer value in doubleing-point.
extern "C"
__global__ void vec_round (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = round(x[id]);
    }
}


// Calculate the reciprocal of the square root of the input argument.
extern "C"
__global__ void vec_rsqrt (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = rsqrt(x[id]);
    }
}


// Calculate the sine of the input argument.
extern "C"
__global__ void vec_sin (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = sin(x[id]);
    }
}


// Calculate the hyperbolic sine of the input argument.
extern "C"
__global__ void vec_sinh (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = sinh(x[id]);
    }
}


// Calculate the sine of the input argument � p .
extern "C"
__global__ void vec_sinpi (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = sinpi(x[id]);
    }
}


// Calculate the square root of the input argument.
extern "C"
__global__ void vec_sqrt (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = sqrt(x[id]);
    }
}


// Calculate the tangent of the input argument.
extern "C"
__global__ void vec_tan (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = tan(x[id]);
    }
}


// Calculate the hyperbolic tangent of the input argument.
extern "C"
__global__ void vec_tanh (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = tanh(x[id]);
    }
}


// Calculate the gamma function of the input argument.
extern "C"
__global__ void vec_tgamma (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = tgamma(x[id]);
    }
}


// Truncate input argument to the integral part.
extern "C"
__global__ void vec_trunc (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = trunc(x[id]);
    }
}


// Calculate the value of the Bessel function of the second kind of order 0 for the input argument.
extern "C"
__global__ void vec_y0 (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = y0(x[id]);
    }
}


// Calculate the value of the Bessel function of the second kind of order 1 for the input argument.
extern "C"
__global__ void vec_y1 (size_t n, double *result, double  *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = y1(x[id]);
    }
}











//=== Vector math (two arguments) ============================================





// Create value with given magnitude, copying sign of second value.
extern "C"
__global__ void vec_copysign (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = copysign(x[id], y[id]);
    }
}

// Compute the positive difference between x and y.
extern "C"
__global__ void vec_fdim (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = fdim(x[id], y[id]);
    }
}

// Divide two doubleing point values.
extern "C"
__global__ void vec_fdivide (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = fdivide(x[id], y[id]);
    }
}

// Determine the maximum numeric value of the arguments.
extern "C"
__global__ void vec_fmax (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = fmax(x[id], y[id]);
    }
}

// Determine the minimum numeric value of the arguments.
extern "C"
__global__ void vec_fmin (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = fmin(x[id], y[id]);
    }
}

// Calculate the doubleing-point remainder of x / y.
extern "C"
__global__ void vec_fmod (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = fmod(x[id], y[id]);
    }
}

// Calculate the square root of the sum of squares of two arguments.
extern "C"
__global__ void vec_hypot (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = hypot(x[id], y[id]);
    }
}

// Return next representable single-precision doubleing-point value afer argument.
extern "C"
__global__ void vec_nextafter (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = nextafter(x[id], y[id]);
    }
}

// Calculate the value of first argument to the power of second argument.
extern "C"
__global__ void vec_pow (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = pow(x[id], y[id]);
    }
}

// Compute single-precision doubleing-point remainder.
extern "C"
__global__ void vec_remainder (size_t n, double *result, double  *x, double  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = remainder(x[id], y[id]);
    }
}




